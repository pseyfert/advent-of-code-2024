#include "cuda_interface.h"
#include "claw.cuh"
#include "cub/util_debug.cuh"
#include <cstdio>
#include "thrust/reduce.h"
#include "thrust/execution_policy.h"
#include "thrust/functional.h"

#include <assert.h>
#define cdpErrchk(ans) \
  { cdpAssert((ans), __FILE__, __LINE__); }
__device__ void cdpAssert(
    hipError_t code, const char* file, int line, bool abort = true) {
  if (code != hipSuccess) {
    printf(
        "%s:%d GPU kernel assert %d: %s \n", file, line, code,
        hipGetErrorString(code));
    if (abort)
      assert(0);
  }
}
#define CubDebugExitPrint(ans) \
  { CubDebugExitPrintImpl((ans), __FILE__, __LINE__); }
__host__ void CubDebugExitPrintImpl(
    hipError_t code, const char* file, int line, bool abort = true) {
  if (code != hipSuccess) {
    printf(
        "%s:%d GPU kernel assert %d: %s \n", file, line, code,
        hipGetErrorString(code));
    if (abort)
      CubDebugExit(code);
  }
}

int pass(int* Ax, int* Bx, int* Ay, int* By, int* Tx, int* Ty, int size) {

  // printf("okay let's do some work for size %d\n", size);

  int* dev_ax;
  int* dev_ay;
  int* dev_bx;
  int* dev_by;
  int* dev_tx;
  int* dev_ty;
  int* res;
  
  CubDebugExit(hipMalloc(&dev_ax, size * sizeof(int)));
  CubDebugExit(hipMalloc(&dev_ay, size * sizeof(int)));
  CubDebugExit(hipMalloc(&dev_bx, size * sizeof(int)));
  CubDebugExit(hipMalloc(&dev_by, size * sizeof(int)));
  CubDebugExit(hipMalloc(&dev_tx, size * sizeof(int)));
  CubDebugExit(hipMalloc(&dev_ty, size * sizeof(int)));
  CubDebugExit(hipMalloc(&res, size * sizeof(int)));

  // printf("init half done\n");

  CubDebugExit(hipMemcpy( dev_ax, Ax, size * sizeof(int), hipMemcpyHostToDevice));
  CubDebugExit(hipMemcpy( dev_ay, Ay, size * sizeof(int), hipMemcpyHostToDevice));
  CubDebugExit(hipMemcpy( dev_bx, Bx, size * sizeof(int), hipMemcpyHostToDevice));
  CubDebugExit(hipMemcpy( dev_by, By, size * sizeof(int), hipMemcpyHostToDevice));
  CubDebugExit(hipMemcpy( dev_tx, Tx, size * sizeof(int), hipMemcpyHostToDevice));
  CubDebugExit(hipMemcpy( dev_ty, Ty, size * sizeof(int), hipMemcpyHostToDevice));
  
  // printf("init done\n");

  run_claw<<<1, size>>>(dev_ax, dev_bx, dev_ay, dev_by, dev_tx, dev_ty, res);

  // printf("call done\n");

  CubDebugExitPrint(hipPeekAtLastError());
  CubDebugExitPrint(hipDeviceSynchronize());
  CubDebugExitPrint(hipPeekAtLastError());

  // printf("sync done\n");

  auto t = thrust::reduce(thrust::device, res, res+size, 0, thrust::plus<int>());

  CubDebugExitPrint(hipFree(dev_ax));
  CubDebugExitPrint(hipFree(dev_ay));
  CubDebugExitPrint(hipFree(dev_bx));
  CubDebugExitPrint(hipFree(dev_by));
  CubDebugExitPrint(hipFree(dev_tx));
  CubDebugExitPrint(hipFree(dev_ty));
  CubDebugExitPrint(hipFree(res));


  return t;
}
